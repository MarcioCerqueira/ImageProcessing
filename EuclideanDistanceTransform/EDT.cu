#include "hip/hip_runtime.h"
#include "EDT.h"

__device__ float GPUComputeEuclideanDistance(int sitePixel, int pixel, int cols) {

	int sx = sitePixel % cols;
	int sy = sitePixel / cols;
	int px = pixel % cols;
	int py = pixel / cols;
	return sqrtf((sx - px) * (sx - px) + (sy - py) * (sy - py));

}

__device__ bool GPUHasDomination(int a, int b, int c, int column, int cols) {

	float u, v;
	//p(i, u)
	int ax = a % cols;
	int ay = a / cols;
	int bx = b % cols;
	int by = b / cols;
	float mx = (float)(ax + bx) / 2;
	float my = (float)(ay + by) / 2;
	if(bx == ax) {
		u = my;
	} else if(by == ay) {
		u = my;
	} else {
		float m1 = (float)(by - ay) / (float)(bx - ax);
		float m2 = -1/m1;
		u = m2 * (column - mx) + my;
	}

	//q(i, v)
	int cx = c % cols;
	int cy = c / cols;
	mx = (float)(bx + cx) / 2;
	my = (float)(by + cy) / 2;
	if(cx == bx) {
		v = my;
	} else if(cy == by) {
		v = my;
	} else {
		float m1 = (float)(cy - by) / (float)(cx - bx);
		float m2 = -1/m1;
		v = m2 * (column - mx) + my;
	}
	
	if(u > v) return true;
	else return false;

}

__global__ void clearStructure(int *structure) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	structure[id] = -1;

}

__global__ void computeSelfNearestSite(unsigned char *image, int *nearestSite) {
	
	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	if(image[pixel] == 0) nearestSite[pixel] = pixel;

}

__global__ void computeNearestSiteInRow(unsigned char *image, int *nearestSite, int cols, int bandSize) {
	
	int x = blockIdx.x * bandSize;
	int y = threadIdx.x;
	for(int xp = x; xp < x + bandSize; xp++) {
	
		int pixel = y * cols + xp;
		if(image[pixel] == 0) {

			for(int xs = xp + 1; xs < x + bandSize; xs++) {
			
				int propagationPixel = y * cols + xs;
				if(image[propagationPixel] != 0) nearestSite[propagationPixel] = pixel;
				else break;
	
			}

		}
	
	}

	for(int xp = x + bandSize - 1; xp >= x; xp--) {
	
		int pixel = y * cols + xp;
		if(image[pixel] == 0) {

			for(int xs = xp - 1; xs >= x; xs--) {
			
				int propagationPixel = y * cols + xs;
				int imagePropagationPixel = image[propagationPixel];
				int nearestSitePropagationPixel = nearestSite[propagationPixel];
				if(nearestSitePropagationPixel == -1) {
					nearestSite[propagationPixel] = pixel;
				} else if(imagePropagationPixel != 0 && nearestSitePropagationPixel != -1) {
					float a = abs(nearestSitePropagationPixel % cols - xs);
					float b = abs(xp - xs);
					if(b < a) nearestSite[propagationPixel] = pixel;
				} else break;

			}

		}

	}
	
}

__global__ void updateBandSitesInRow(int *nearestSite, int cols, int bandSize, int iteration) {

	int x = blockIdx.x * bandSize + ((iteration + 1) % 2) * (bandSize - 1);
	int y = threadIdx.x;
	int pixel = y * cols + x;
	int neighbourPixel;

	if(iteration % 2 == 0) neighbourPixel = pixel + 1;
	else neighbourPixel = pixel + bandSize - 1;

	int nearestPixel1 = nearestSite[pixel];
	int nearestPixel2 = nearestSite[neighbourPixel];
	float a = GPUComputeEuclideanDistance(pixel, nearestPixel1, cols);
	float b = GPUComputeEuclideanDistance(pixel, nearestPixel2, cols);
	float c = GPUComputeEuclideanDistance(neighbourPixel, nearestPixel1, cols);
	float d = GPUComputeEuclideanDistance(neighbourPixel, nearestPixel2, cols);
	if(a > b) nearestSite[pixel] = nearestPixel2;
	if(d > c) nearestSite[neighbourPixel] = nearestPixel1;
	
}

__global__ void updateNearestSiteInRow(int *nearestSite, int cols, int bandSize) {
	
	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	int x = pixel % cols;
	int y = pixel / cols;
	int band = x / bandSize;
	int firstBandPixel = y * cols + band * bandSize;
	int lastBandPixel = y * cols + band * bandSize + bandSize - 1;
	float a = GPUComputeEuclideanDistance(pixel, nearestSite[pixel], cols);
	float b = GPUComputeEuclideanDistance(pixel, nearestSite[firstBandPixel], cols);
	float c = GPUComputeEuclideanDistance(pixel, nearestSite[lastBandPixel], cols);
	if(b < a && b <= c) nearestSite[pixel] = nearestSite[firstBandPixel];
	if(c < b && c < a) nearestSite[pixel] = nearestSite[lastBandPixel];
	
}

__global__ void computeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols, int bandSize) {

	//Here, our stack begins in "y + bandSize - 1" and ends in "y"
	int x = threadIdx.x;
	int y = blockIdx.x * bandSize;
	int count = y;

	for(int yb = y; yb < y + bandSize; yb++) {

		int pixel = yb * cols + x;
		int c = nearestSite[pixel];
		if(c != -1) {

			while(count >= y + 2) {
					
				int a = proximateSites[(count - 2) * cols + x];
				int b = proximateSites[(count - 1) * cols + x];
				if(GPUHasDomination(a, b, c, x, cols)) {
					proximateSites[(count - 1) * cols + x] = -1;
					count--;
				} else break;

			}
				
			proximateSites[count * cols + x] = c;
			count++;

		}

	}

}

__global__ void mergeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols, int bandSize) {

	int x = threadIdx.x;
	int count = 0;
	
	for(int y = 0; y < bandSize; y++)
		if(proximateSites[y * cols + x] != -1) count++;	
	
	for(int it = 1; it < rows/bandSize; it++) {
		int bandCount = 0;
		for(int y = 0; y < bandSize; y++) {

			int yp = y + it * bandSize;
			int pixel = yp * cols + x;
			int c = proximateSites[pixel];
			if(c != -1) {
			
				if(bandCount == 2) {
					proximateSites[count * cols + x] = c;
					count++;
					continue;
				}

				while(count >= 2) {
					
					int a = proximateSites[(count - 2) * cols + x];
					int b = proximateSites[(count - 1) * cols + x];
					if(GPUHasDomination(a, b, c, x, cols)) {
						proximateSites[(count - 1) * cols + x] = -1;
						count--;
						bandCount = 0;
					} else break;

				}
			
				proximateSites[count * cols + x] = c;
				count++;
				bandCount++;
				
			}

		}
	}

}

__global__ void computeNearestSiteInFullKernel(int *proximateSites, int *nearestSite, int rows, int cols) {

	int x = threadIdx.x;
	int count = 0;

	for(int y = 0; y < rows; y++) {

		int pixel = y * cols + x;
		while(count < rows - 1) {

			float a = GPUComputeEuclideanDistance(proximateSites[count * cols + x], pixel, cols);
			float b = GPUComputeEuclideanDistance(proximateSites[(count + 1) * cols + x], pixel, cols);
			if(a <= b) break;
			else count++;
				
				
		}
			
		nearestSite[pixel] = proximateSites[count * cols + x];
			
	}
	
}

__global__ void computeDistanceTransform(unsigned char *EDTImage, int *nearestSite, int cols) {
	
	int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	EDTImage[pixel] = GPUComputeEuclideanDistance(nearestSite[pixel], pixel, cols);

}

void GPUCheckError(char *methodName) {

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess) printf("%s: %s\n", methodName, hipGetErrorString(error));
	
}

void GPUClearStructure(int *structure, int rows, int cols) {

    clearStructure<<<rows, cols>>>(structure);
	GPUCheckError("GPUClearStructure");

}

void GPUComputeNearestSiteInRow(unsigned char *image, int *nearestSite, int rows, int cols) {

	int bands = 16;
	int bandSize = cols / bands;
	computeSelfNearestSite<<<rows, cols>>>(image, nearestSite);
	computeNearestSiteInRow<<<bands, rows>>>(image, nearestSite, cols, bandSize);
	for(int it = 0; it < bands; it++) updateBandSitesInRow<<<bands - (int)((it + 1) % 2), rows>>>(nearestSite, cols, bandSize, it);
	updateNearestSiteInRow<<<rows, cols>>>(nearestSite, cols, bandSize);
	GPUCheckError("GPUComputeNearestSiteInRow");

}

void GPUComputeProximateSitesInColumn(int *nearestSite, int *proximateSites, int rows, int cols) {

	int bands = 16;
	int bandSize = rows / bands;
	computeProximateSitesInColumn<<<bands, cols>>>(nearestSite, proximateSites, rows, cols, bandSize);
	mergeProximateSitesInColumn<<<1, cols>>>(nearestSite, proximateSites, rows, cols, bandSize);
	GPUCheckError("GPUComputeProximateSitesInColumn");

}

void GPUComputeNearestSiteInFull(int *proximateSites, int *nearestSite, int rows, int cols) {

	computeNearestSiteInFullKernel<<<1, cols>>>(proximateSites, nearestSite, rows, cols);
	GPUCheckError("GPUComputeNearestSiteInFull");
	
}

void GPUComputeDistanceTransform(unsigned char *EDTImage, int *nearestSite, int rows, int cols) {

	computeDistanceTransform<<<rows, cols>>>(EDTImage, nearestSite, cols);
	GPUCheckError("GPUComputeDistanceTransform");

}
